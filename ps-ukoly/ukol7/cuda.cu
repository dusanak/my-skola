#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "md5.cu"

#define ALPHABET_SIZE 26

//prevede cislo na ekvivalentni retezec
__device__ void numberToString(uint64_t input_number, uint8_t ** output_string, int string_length) {   
    uint8_t tmp[40];
    tmp[string_length] = '\0';
    for (int i = string_length - 1; i >= 0; i--) {
        tmp[i] = 'a' + (input_number % ALPHABET_SIZE);
        input_number = input_number / ALPHABET_SIZE;
    }

    *output_string = tmp;
}

//prevod retezce na vstupu do MD5
__device__ void convertStringToMD5(uint64_t idx, int string_length) {
    uint8_t md_value[16];
    uint8_t * input_string;

    numberToString(idx, &input_string, string_length);
    md5(reinterpret_cast<const uint8_t *>(input_string), string_length, md_value);
    
    printf("%2.2x%2.2x%2.2x%2.2x%2.2x%2.2x%2.2x%2.2x%2.2x%2.2x%2.2x%2.2x%2.2x%2.2x%2.2x%2.2x ",
     md_value[0], md_value[1], md_value[2], md_value[3], md_value[4], md_value[5], 
     md_value[6], md_value[7], md_value[8], md_value[9], md_value[10], md_value[11], 
     md_value[12], md_value[13], md_value[14], md_value[15]);
}

__global__ void convertToMD5(uint64_t number_of_strings, int string_length) {
    //vypocitava ciselne hodnoty jednotlivych retezcu ktere jsou nasledne prepocitavany na samotne retezce
    uint64_t idx = (blockIdx.x * blockDim.x + (threadIdx.x));

    if (idx >= number_of_strings)
        return;

    convertStringToMD5(idx, string_length);
}

//generuje MD5 hashe vsech retezcu v abecede a-z pro zadanou string_length
//kazde vlakno zpracova prave jeden retezec
void generateMD5Cuda(int string_length, int number_of_threads) {
	hipError_t cerr;

    uint64_t number_of_strings = (uint64_t)(pow((double)ALPHABET_SIZE, (double)string_length) + 0.5);

    convertToMD5<<< (number_of_strings / number_of_threads) + 1, number_of_threads >>>(number_of_strings, string_length);

	if ( ( cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	// Output from printf is in GPU memory. 
	// To get its contens it is necessary to synchronize device.

	hipDeviceSynchronize();
}