#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "md5.cu"

#define ALPHABET_SIZE 26
#define NUMBER_OF_THREADS 128

//prevede cislo na ekvivalentni retezec
__device__ void numberToString(int input_number, uint8_t ** output_string, int string_length) {   
    uint8_t tmp[16];
    tmp[string_length] = '\0';
    for (int i = string_length - 1; i >= 0; i--) {
        tmp[i] = 'a' + (input_number % ALPHABET_SIZE);
        input_number = input_number / ALPHABET_SIZE;
    }

    *output_string = tmp;
}

//prevod retezce na vstupu do MD5
__device__ void convertStringToMD5(int idx, int string_length) {
    uint8_t md_value[16];
    uint8_t * input_string;

    numberToString(idx, &input_string, string_length);

    //printf(reinterpret_cast<char *>(input_string));

    md5(reinterpret_cast<const uint8_t *>(input_string), string_length, md_value);

    //printf("%2.2x%2.2x%2.2x%2.2x\n", output_string[3], output_string[2], output_string[1], output_string[0]);
    
    printf("%2.2x%2.2x%2.2x%2.2x%2.2x%2.2x%2.2x%2.2x%2.2x%2.2x%2.2x%2.2x%2.2x%2.2x%2.2x%2.2x ",
     md_value[0], md_value[1], md_value[2], md_value[3], md_value[4], md_value[5], 
     md_value[6], md_value[7], md_value[8], md_value[9], md_value[10], md_value[11], 
     md_value[12], md_value[13], md_value[14], md_value[15]);
}

__global__ void convertToMD5(int number_of_strings, int string_length) {
    uint32_t idx = (blockIdx.x * blockDim.x + threadIdx.x);

    if (idx >= number_of_strings)
        return;

    convertStringToMD5(idx, string_length);
}

void generateMD5Cuda(int string_length) {
	hipError_t cerr;
	// Following command can increase internal buffer for printf function
    /*cerr = hipDeviceSetLimit( hipLimitPrintfFifoSize, required_size );
	if ( err != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );
    */

	// Thread creation from selected kernel:
	// first parameter dim3 is grid dimension
	// second parameter dim3 is block dimension

    int number_of_strings = (int)(pow((double)ALPHABET_SIZE, (double)string_length) + 0.5);
    convertToMD5<<< (number_of_strings / NUMBER_OF_THREADS) + 1, NUMBER_OF_THREADS >>>(number_of_strings, string_length);

	if ( ( cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	// Output from printf is in GPU memory. 
	// To get its contens it is necessary to synchronize device.

	hipDeviceSynchronize();
}